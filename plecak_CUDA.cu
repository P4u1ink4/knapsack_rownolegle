
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define MAX_VALUE 10

__global__ void dynamic_kernel(int bag, int *items_weight, int *items_val, int n, int *result, int *chosen_items) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n) {
        extern __shared__ int temp[]; // Shared memory for dynamic programming table
        int *matrix = temp + (bag + 1) * idx; // Each thread has its own row in the matrix

        for (int j = 0; j <= bag; j++) {
            if (idx == 0 || j == 0)
                matrix[j] = 0;
            else if (items_weight[idx] <= j) {
                int newVal = matrix[j - items_weight[idx]] + items_val[idx];
                matrix[j] = (newVal > matrix[j]) ? newVal : matrix[j];
            }
            else
                matrix[j] = matrix[j];

            __syncthreads(); // Synchronize threads after each j iteration
        }

        result[idx] = matrix[bag];

        // Record chosen items
        if (matrix[bag] != 0)
            chosen_items[idx] = 1;
    }
}

void print_chosen_items(int n, int *items_weight, int *items_val, int *chosen_items) {
    printf("Przedmioty w plecaku:\n");
    for (int i = 0; i < n; i++) {
        if (chosen_items[i] == 1) {
            printf("  Przedmiot %d: Waga: %d, Wartosc: %d\n", i, items_weight[i], items_val[i]);
        }
    }
}

int dynamic_cuda(int bag, int *items_weight, int *items_val, int n) {
    int *d_items_weight, *d_items_val, *d_result, *d_chosen_items;
    int *result = (int *)malloc(n * sizeof(int));
    int *chosen_items = (int *)malloc(n * sizeof(int));

    hipMalloc(&d_items_weight, n * sizeof(int));
    hipMalloc(&d_items_val, n * sizeof(int));
    hipMalloc(&d_result, n * sizeof(int));
    hipMalloc(&d_chosen_items, n * sizeof(int));

    hipMemcpy(d_items_weight, items_weight, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_items_val, items_val, n * sizeof(int), hipMemcpyHostToDevice);

    dynamic_kernel<<<(n + 255) / 256, 256, (bag + 1) * sizeof(int)>>>(bag, d_items_weight, d_items_val, n, d_result, d_chosen_items);

    hipMemcpy(result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(chosen_items, d_chosen_items, n * sizeof(int), hipMemcpyDeviceToHost);

    int max_result = 0;
    for (int i = 0; i < n; i++) {
        if (result[i] > max_result)
            max_result = result[i];
    }

    print_chosen_items(n, items_weight, items_val, chosen_items);

    free(result);
    free(chosen_items);
    hipFree(d_items_weight);
    hipFree(d_items_val);
    hipFree(d_result);
    hipFree(d_chosen_items);

    return max_result;
}


void separator(int n, int bag, int *items_weight, int *items_val, int *items_priority) {
    int n_elements_with_priority = 0;
    int max_weight_elements_with_priority = 0;
    int n_elements_without_priority = 0;
    int *first_use_elements_weight = (int *)malloc(n * sizeof(int));
    int *first_use_elements_val = (int *)malloc(n * sizeof(int));
    int *else_elements_weight = (int *)malloc(n * sizeof(int));
    int *else_elements_val = (int *)malloc(n * sizeof(int));
    int max_sum_dynamic = 0;

    for (int i = 0; i < n; i++) {
        if (items_priority[i] == 1) {
            first_use_elements_weight[n_elements_with_priority] = items_weight[i];
            first_use_elements_val[n_elements_with_priority] = items_val[i];
            n_elements_with_priority++;
            max_weight_elements_with_priority += items_weight[i];
            max_sum_dynamic += items_val[i];
        } else {
            else_elements_weight[n_elements_without_priority] = items_weight[i];
            else_elements_val[n_elements_without_priority] = items_val[i];
            n_elements_without_priority++;
        }
    }

    if (max_weight_elements_with_priority <= bag) {
        printf("Przedmioty konieczne w plecaku\n");
        bag -= max_weight_elements_with_priority;
        max_sum_dynamic += dynamic_cuda(bag, else_elements_weight, else_elements_val, n_elements_without_priority);
    } else {
        printf("Nie udalo zmiescic sie wszystkich koniecznych przedmiotow w plecaku.\n");
        max_sum_dynamic = dynamic_cuda(bag, items_weight, items_val, n);
    }

    printf("Wynik algorytmu dynamicznego: %d\n", max_sum_dynamic);

    free(first_use_elements_weight);
    free(first_use_elements_val);
    free(else_elements_weight);
    free(else_elements_val);
}

void generator(int n, int bag, int *items_weight, int *items_val, int *items_priority) {
    double start, end;

    start = (double) clock();
    start = start / CLOCKS_PER_SEC;

    separator(n, bag, items_weight, items_val, items_priority);

    end = (double) clock();
    end = end / CLOCKS_PER_SEC;

    printf("Czas wykonania: %f s\n", end - start);
}

int main(int argc, char *argv[]) {
    // int n_items[6] = {5000,10000,15000,20000,25000,30000};
    // int bag_sizes[6] = {10000,10000,10000,10000,10000,10000};

    int n_items = 3;
    int bag_size = 4;
    int items_weight[3] = {2,3,2};
    int items_val[3] = {3,4,2};
    int items_priority[3] = {0,0,0};

    generator(n_items, bag_size, items_weight, items_val, items_priority);

    free(items_weight);
    free(items_val);
    free(items_priority);

    return 0;
}
