#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define MAX_VALUE 10

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void dynamic_kernel(int bag, int *items_weight, int *items_val, int n, int *result, int *matrix) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx <= bag) {
        for (int i = 0; i <= n; i++) {
            __syncthreads();
            int currentRow = i % 2;
            int previousRow = (i - 1) % 2;
            if (idx == 0) {
                matrix[(bag+1)*currentRow] = 0;
            }
            else if( i==0 ){
                matrix[idx] = 0;
            }
            else if (idx >= items_weight[i - 1]) {
                int val = matrix[(bag+1)*previousRow + idx - items_weight[i - 1]] + items_val[i - 1];
                matrix[(bag+1)*currentRow + idx] = max(matrix[(bag+1)*previousRow + idx], val);
            }
            else {
                matrix[(bag+1)*currentRow + idx] = matrix[(bag+1)*previousRow + idx];
            }
            __syncthreads();
        }
        if (idx == bag) {
            atomicMax(result, matrix[(bag+1)*(n % 2) + idx]);
        }
    }
}

int dynamic_cuda(int bag, int *items_weight, int *items_val, int n) {
    int *d_items_weight, *d_items_val, *d_result;

    hipMalloc((void **)&d_items_weight, n * sizeof(int));
    hipMalloc((void **)&d_items_val, n * sizeof(int));
    hipMalloc((void **)&d_result, sizeof(int));

    hipMemcpy(d_items_weight, items_weight, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_items_val, items_val, n * sizeof(int), hipMemcpyHostToDevice);

    int *d_matrix;
    hipMalloc((void **)&d_matrix, (bag+1) * 2 * sizeof(int));

    int threadsPerBlock = 256;
    int blocksPerGrid = (bag + threadsPerBlock - 1) / threadsPerBlock;

    dynamic_kernel<<<blocksPerGrid, threadsPerBlock>>>(bag, d_items_weight, d_items_val, n, d_result, d_matrix);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    int result;
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_items_weight);
    hipFree(d_items_val);
    hipFree(d_result);

    return result;
}

void separator(int n, int bag, int *items_weight, int *items_val, int *items_priority) {
    int n_elements_with_priority = 0;
    int max_weight_elements_with_priority = 0;
    int n_elements_without_priority = 0;
    int *first_use_elements_weight = (int *)malloc(n * sizeof(int));
    int *first_use_elements_val = (int *)malloc(n * sizeof(int));
    int *else_elements_weight = (int *)malloc(n * sizeof(int));
    int *else_elements_val = (int *)malloc(n * sizeof(int));
    int max_sum_dynamic = 0;

    for (int i = 0; i < n; i++) {
        if (items_priority[i] == 1) {
            first_use_elements_weight[n_elements_with_priority] = items_weight[i];
            first_use_elements_val[n_elements_with_priority] = items_val[i];
            n_elements_with_priority++;
            max_weight_elements_with_priority += items_weight[i];
            max_sum_dynamic += items_val[i];
        } else {
            else_elements_weight[n_elements_without_priority] = items_weight[i];
            else_elements_val[n_elements_without_priority] = items_val[i];
            n_elements_without_priority++;
        }
    }

    if (max_weight_elements_with_priority <= bag) {
        printf("Przedmioty konieczne w plecaku\n");
        bag -= max_weight_elements_with_priority;
        if(bag>0){
            max_sum_dynamic += dynamic_cuda(bag, else_elements_weight, else_elements_val, n_elements_without_priority);
        }
    } else {
        printf("Nie udalo sie zmiescic wszystkich koniecznych przedmioow w plecaku.\n");
        max_sum_dynamic = dynamic_cuda(bag, items_weight, items_val, n);
    }

    printf("Wynik algorytmu dynamicznego: %d\n", max_sum_dynamic);

    free(first_use_elements_weight);
    free(first_use_elements_val);
    free(else_elements_weight);
    free(else_elements_val);
}

void generator(int n, int bag, int *items_weight, int *items_val, int *items_priority) {
    double start, end;

    start = (double) clock();
    start = start / CLOCKS_PER_SEC;

    separator(n, bag, items_weight, items_val, items_priority);

    end = (double) clock();
    end = end / CLOCKS_PER_SEC;

    printf("Czas wykonania: %f s\n", end - start);
}

int main(int argc, char *argv[]) {

    int n_items = atoi(argv[1]);
    int bag_size = atoi(argv[2]);

    int *items_weight = (int *)malloc(n_items * sizeof(int));
    int *items_val = (int *)malloc(n_items * sizeof(int));
    int *items_priority = (int *)malloc(n_items * sizeof(int));

    for (int j = 0; j < n_items; j++) {
        items_weight[j] = rand() % (bag_size / 2) + 1;
        items_val[j] = rand() % MAX_VALUE + 1;
        items_priority[j] = 0;
        // items_priority[j] = rand() % (n_items / 2 ) == 0 ? 1 : 0;
    }

    generator(n_items, bag_size, items_weight, items_val, items_priority);

    free(items_weight);
    free(items_val);
    free(items_priority);

    return 0;
}
