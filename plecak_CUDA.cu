
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define MAX_VALUE 10

__global__ void dynamic_kernel(int bag, int *items_weight, int *items_val, int n, int *result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n) {
        extern __shared__ int temp[]; // Shared memory for dynamic programming table
        int *matrix = temp;

        for (int j = 0; j <= bag; j++) {
            if (idx == 0 || j == 0)
                matrix[j] = 0;
            else if (items_weight[idx - 1] <= j) {
                int newVal = matrix[j - items_weight[idx - 1]] + items_val[idx - 1];
                matrix[j] = (newVal > matrix[j]) ? newVal : matrix[j];
            }
            else
                matrix[j] = matrix[j];
        }

        result[idx] = matrix[bag];
    }
}


int dynamic_cuda(int bag, int *items_weight, int *items_val, int n) {
    int *d_items_weight, *d_items_val, *d_result;
    int *result = (int *)malloc(n * sizeof(int));

    hipMalloc(&d_items_weight, n * sizeof(int));
    hipMalloc(&d_items_val, n * sizeof(int));
    hipMalloc(&d_result, n * sizeof(int));

    hipMemcpy(d_items_weight, items_weight, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_items_val, items_val, n * sizeof(int), hipMemcpyHostToDevice);

    dynamic_kernel<<<(n + 255) / 256, 256, (bag + 1) * sizeof(int)>>>(bag, d_items_weight, d_items_val, n, d_result);

    hipMemcpy(result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);

    int last_result = result[n - 1];

    free(result);
    hipFree(d_items_weight);
    hipFree(d_items_val);
    hipFree(d_result);

    return last_result;
}

void separator(int n, int bag, int *items_weight, int *items_val, int *items_priority) {
    int n_elements_with_priority = 0;
    int max_weight_elements_with_priority = 0;
    int n_elements_without_priority = 0;
    int *first_use_elements_weight = (int *)malloc(n * sizeof(int));
    int *first_use_elements_val = (int *)malloc(n * sizeof(int));
    int *else_elements_weight = (int *)malloc(n * sizeof(int));
    int *else_elements_val = (int *)malloc(n * sizeof(int));
    int max_sum_dynamic = 0;

    for (int i = 0; i < n; i++) {
        if (items_priority[i] == 1) {
            first_use_elements_weight[n_elements_with_priority] = items_weight[i];
            first_use_elements_val[n_elements_with_priority] = items_val[i];
            n_elements_with_priority++;
            max_weight_elements_with_priority += items_weight[i];
            max_sum_dynamic += items_val[i];
        } else {
            else_elements_weight[n_elements_without_priority] = items_weight[i];
            else_elements_val[n_elements_without_priority] = items_val[i];
            n_elements_without_priority++;
        }
    }

    if (max_weight_elements_with_priority <= bag) {
        printf("Przedmioty konieczne w plecaku\n");
        bag -= max_weight_elements_with_priority;
        max_sum_dynamic += dynamic_cuda(bag, else_elements_weight, else_elements_val, n_elements_without_priority);
    } else {
        printf("Nie udalo zmiescic sie wszystkich koniecznych przedmiotow w plecaku.\n");
        max_sum_dynamic = dynamic_cuda(bag, items_weight, items_val, n);
    }

    printf("Wynik algorytmu dynamicznego: %d\n", max_sum_dynamic);

    free(first_use_elements_weight);
    free(first_use_elements_val);
    free(else_elements_weight);
    free(else_elements_val);
}

void generator(int n, int bag, int *items_weight, int *items_val, int *items_priority) {
    double start, end;

    start = (double) clock();
    start = start / CLOCKS_PER_SEC;

    separator(n, bag, items_weight, items_val, items_priority);

    end = (double) clock();
    end = end / CLOCKS_PER_SEC;

    printf("Czas wykonania: %f s\n", end - start);
}

int main(int argc, char *argv[]) {
    // int n_items[6] = {5000,10000,15000,20000,25000,30000};
    // int bag_sizes[6] = {10000,10000,10000,10000,10000,10000};

    int n_items = atoi(argv[1]);
    int bag_size = atoi(argv[2]);

    int *items_weight = (int *)malloc(n_items * sizeof(int));
    int *items_val = (int *)malloc(n_items * sizeof(int));
    int *items_priority = (int *)malloc(n_items * sizeof(int));

    for (int j = 0; j < n_items; j++) {
        items_weight[j] = rand() % (bag_size / 2) + 1;
        items_val[j] = rand() % MAX_VALUE + 1;
        // items_priority[j] = 0;
        items_priority[j] = rand() % (n_items / 2 ) == 0 ? 1 : 0;
    }

    generator(n_items, bag_size, items_weight, items_val, items_priority);

    free(items_weight);
    free(items_val);
    free(items_priority);

    return 0;
}
