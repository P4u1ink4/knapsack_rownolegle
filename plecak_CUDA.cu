#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define MAX_VALUE 10

__global__ void dynamic_kernel(int bag, int *items_weight, int *items_val, int n, int *result) {
    int idx = threadIdx.x;
    extern __shared__ int matrix[];

    matrix[idx] = 0;
    __syncthreads();

    for (int i = 1; i <= n; i++) {
        for (int j = bag; j >= items_weight[i - 1]; j--) {
            if (idx >= items_weight[i - 1])
                atomicMax(&matrix[j], matrix[j - items_weight[i - 1]] + items_val[i - 1]);
        }
        __syncthreads();
    }

    if (idx == 0) {
        for (int i = 1; i <= bag; i++) {
            atomicMax(result, matrix[i]);
        }
    }
}

int dynamic_cuda(int bag, int *items_weight, int *items_val, int n) {
    int *d_items_weight, *d_items_val, *d_result;

    hipMalloc((void **)&d_items_weight, n * sizeof(int));
    hipMalloc((void **)&d_items_val, n * sizeof(int));
    hipMalloc((void **)&d_result, sizeof(int));

    hipMemcpy(d_items_weight, items_weight, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_items_val, items_val, n * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    int blocksPerGrid = (bag + threadsPerBlock - 1) / threadsPerBlock;

    dynamic_kernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(bag, d_items_weight, d_items_val, n, d_result);

    int result;
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_items_weight);
    hipFree(d_items_val);
    hipFree(d_result);

    return result;
}

void separator(int n, int bag, int *items_weight, int *items_val, int *items_priority) {
    int n_elements_with_priority = 0;
    int max_weight_elements_with_priority = 0;
    int n_elements_without_priority = 0;
    int *first_use_elements_weight = (int *)malloc(n * sizeof(int));
    int *first_use_elements_val = (int *)malloc(n * sizeof(int));
    int *else_elements_weight = (int *)malloc(n * sizeof(int));
    int *else_elements_val = (int *)malloc(n * sizeof(int));
    int max_sum_dynamic = 0;

    for (int i = 0; i < n; i++) {
        if (items_priority[i] == 1) {
            first_use_elements_weight[n_elements_with_priority] = items_weight[i];
            first_use_elements_val[n_elements_with_priority] = items_val[i];
            n_elements_with_priority++;
            max_weight_elements_with_priority += items_weight[i];
            max_sum_dynamic += items_val[i];
        } else {
            else_elements_weight[n_elements_without_priority] = items_weight[i];
            else_elements_val[n_elements_without_priority] = items_val[i];
            n_elements_without_priority++;
        }
    }

    if (max_weight_elements_with_priority <= bag) {
        printf("Przedmioty konieczne w plecaku\n");
        bag -= max_weight_elements_with_priority;
        max_sum_dynamic += dynamic_cuda(bag, else_elements_weight, else_elements_val, n_elements_without_priority);
    } else {
        printf("Nie udało się zmieścić wszystkich koniecznych przedmiotów w plecaku.\n");
        max_sum_dynamic = dynamic_cuda(bag, items_weight, items_val, n);
    }

    printf("Wynik algorytmu dynamicznego: %d\n", max_sum_dynamic);

    free(first_use_elements_weight);
    free(first_use_elements_val);
    free(else_elements_weight);
    free(else_elements_val);
}

void generator(int n, int bag, int *items_weight, int *items_val, int *items_priority) {
    double start, end;

    start = (double) clock();
    start = start / CLOCKS_PER_SEC;

    separator(n, bag, items_weight, items_val, items_priority);

    end = (double) clock();
    end = end / CLOCKS_PER_SEC;

    printf("Czas wykonania: %f s\n", end - start);
}

int main(int argc, char *argv[]) {

    int n_items = atoi(argv[1]);
    int bag_size = atoi(argv[2]);

    int *items_weight = (int *)malloc(n_items * sizeof(int));
    int *items_val = (int *)malloc(n_items * sizeof(int));
    int *items_priority = (int *)malloc(n_items * sizeof(int));

    for (int j = 0; j < n_items; j++) {
        items_weight[j] = rand() % (bag_size / 2) + 1;
        items_val[j] = rand() % MAX_VALUE + 1;
        // items_priority[j] = 0;
        items_priority[j] = rand() % (n_items / 2 ) == 0 ? 1 : 0;
    }

    generator(n_items, bag_size, items_weight, items_val, items_priority);

    free(items_weight);
    free(items_val);
    free(items_priority);

    return 0;
}
