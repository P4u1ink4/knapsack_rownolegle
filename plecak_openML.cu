
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <omp.h>

#define MAX_VALUE 10

int dynamic(int bag, int items_weight[], int items_val[], int n);
void separator(int n, int bag, int items_weight[], int items_val[], int items_priority[]);
void generator(int n, int bag, int items_weight[], int items_val[], int items_priority[]);

int main(int argc, char* argv[]) {
    // int n_items[6] = {5000,10000,15000,20000,25000,30000};
    // int bag_sizes[6] = {10000,10000,10000,10000,10000,10000};

    int N = 1;
    if (argc > 1)  N = atoi(argv[1]);

    omp_set_num_threads(N);

    int n_items = atoi(argv[2]);
    int bag_size = atoi(argv[3]);

    int *items_weight = (int *)malloc(n_items * sizeof(int));
    int *items_val = (int *)malloc(n_items * sizeof(int));
    int *items_priority = (int *)malloc(n_items * sizeof(int));

    for (int j = 0; j < n_items; j++) {
        items_weight[j] = rand() % (bag_size / 2) + 1;
        items_val[j] = rand() % MAX_VALUE + 1;
        items_priority[j] = 0;
        // items_priority[j] =rand() % (n_items / 2 ) == 0 ? 1 : 0;
    }

    generator(n_items,bag_size,items_weight,items_val,items_priority);
    return 0;
}

int dynamic(int bag, int items_weight[], int items_val[], int n) {
    int **matrix = (int **)malloc((n + 1) * sizeof(int *));
    for (int i = 0; i <= n; i++) {
        matrix[i] = (int *)malloc((bag + 1) * sizeof(int));
    }

    for (int i = 0; i <= n; i++) {
        #pragma omp parallel for
        for (int j = 0; j <= bag; j++) {
            if (i == 0 || j == 0)
                matrix[i][j] = 0;
            else if (items_weight[i - 1] <= j)
                matrix[i][j] = fmax(matrix[i - 1][j], matrix[i - 1][j - items_weight[i - 1]] + items_val[i - 1]);
            else
                matrix[i][j] = matrix[i - 1][j];
        }
    }

    for (int i = 0; i <= n; i++) {
        free(matrix[i]);
    }
    free(matrix);

    return matrix[n][bag];
}

void separator(int n, int bag, int items_weight[], int items_val[], int items_priority[]) {
    int n_elements_with_priority = 0;
    int max_weight_elements_with_priority = 0;
    int n_elements_without_priority = 0;
    int *first_use_elements_weight = (int *)malloc(n * sizeof(int));
    int *first_use_elements_val = (int *)malloc(n * sizeof(int));
    int *else_elements_weight = (int *)malloc(n * sizeof(int));
    int *else_elements_val = (int *)malloc(n * sizeof(int));
    int max_sum_dynamic = 0;

    for (int i = 0; i < n; i++) {
        if (items_priority[i] == 1) {
            first_use_elements_weight[n_elements_with_priority] = items_weight[i];
            first_use_elements_val[n_elements_with_priority] = items_val[i];
            n_elements_with_priority++;
            max_weight_elements_with_priority += items_weight[i];
            max_sum_dynamic += items_val[i];
        } else {
            else_elements_weight[n_elements_without_priority] = items_weight[i];
            else_elements_val[n_elements_without_priority] = items_val[i];
            n_elements_without_priority++;
        }
    }

    if (max_weight_elements_with_priority <= bag) {
        printf("Przedmioty konieczne w plecaku\n");
        bag -= max_weight_elements_with_priority;
        max_sum_dynamic += dynamic(bag, else_elements_weight, else_elements_val, n_elements_without_priority);
    } else {
        printf("Nie udalo zmiescic sie wszystkich koniecznych przedmiotow w plecaku.\n");
        max_sum_dynamic = dynamic(bag, items_weight, items_val, n);
    }

    printf("Wynik algorytmu dynamicznego: %d\n", max_sum_dynamic);

    free(first_use_elements_weight);
    free(first_use_elements_val);
    free(else_elements_weight);
    free(else_elements_val);
}

void generator(int n, int bag, int items_weight[], int items_val[], int items_priority[]) {
    double start, end;

    start = omp_get_wtime();
    
    separator(n, bag, items_weight, items_val, items_priority); 
    
    end = omp_get_wtime();
    
    printf("Czas wykonania: %f s\n", end - start);
}
